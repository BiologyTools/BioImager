
#include <hip/hip_runtime.h>
__global__ void copyTileToCanvas(unsigned char* canvas, int canvasWidth, int canvasHeight,
    unsigned char* tile, int tileWidth, int tileHeight,
    int offsetX, int offsetY)
{
    // Calculate the global x and y index for the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the bounds of the tile and canvas
    if (x < tileWidth && y < tileHeight) {
        int canvasX = x + offsetX;
        int canvasY = y + offsetY;

        // Ensure the canvas indices are within bounds
        if (canvasX < canvasWidth && canvasY < canvasHeight) {
            // Calculate the source and destination indices
            int tileIdx = (y * tileWidth + x) * 3;   // Each pixel has 3 components (RGB)
            int canvasIdx = (canvasY * canvasWidth + canvasX) * 3;

            // Copy the pixel (RGB components)
            canvas[canvasIdx] = tile[tileIdx];
            canvas[canvasIdx + 1] = tile[tileIdx + 1];
            canvas[canvasIdx + 2] = tile[tileIdx + 2];
        }
    }
}
