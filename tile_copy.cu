
#include <hip/hip_runtime.h>
__global__ void copyTileToCanvas(
    unsigned char* canvas, int canvasWidth, int canvasHeight,
    unsigned char* tile, int tileWidth, int tileHeight,
    int offsetX, int offsetY,
    int canvasTileWidth, int canvasTileHeight)
{
    // Calculate the global x and y index for the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the bounds of the tile and canvas
    if (x < canvasTileWidth && y < canvasTileHeight) {
        int canvasX = x + offsetX;
        int canvasY = y + offsetY;

        // Ensure the canvas indices are within bounds
        if (canvasX < canvasWidth && canvasY < canvasHeight) {
            // Calculate corresponding tile indices, scaling tile pixels to canvas size
            float scaleX = (float)tileWidth / canvasTileWidth;
            float scaleY = (float)tileHeight / canvasTileHeight;

            // Calculate tile indices by scaling the x, y coordinates
            int tileX = (int)(x * scaleX);
            int tileY = (int)(y * scaleY);

            // Ensure tile indices are within the tile bounds
            if (tileX < tileWidth && tileY < tileHeight) {
                int tileIdx = (tileY * tileWidth + tileX) * 3;   // Each pixel has 3 components (RGB)
                int canvasIdx = (canvasY * canvasWidth + canvasX) * 3;

                // Copy the pixel (RGB components)
                canvas[canvasIdx] = tile[tileIdx];
                canvas[canvasIdx + 1] = tile[tileIdx + 1];
                canvas[canvasIdx + 2] = tile[tileIdx + 2];
            }
        }
    }
}
